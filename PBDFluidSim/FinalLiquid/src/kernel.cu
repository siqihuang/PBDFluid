#include "hip/hip_runtime.h"
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/device_vector.h>
#include <thrust/remove.h>
#include "kernel.h"

static int GparticleNum=0,GprimitiveNum=0;
static glm::vec3 GparticleDim=glm::vec3(0);
static GPUParticle *Gparticles,*dev_particles;
static GPUContainer *Gcontainer,*dev_container;
static GPUCell *Gcell,*dev_cell;
static GPUPrimitive *Gprimitive,*dev_primitive;

__host__ __device__ inline unsigned int utilhash(unsigned int a) {
    a = (a + 0x7ed55d16) + (a << 12);
    a = (a ^ 0xc761c23c) ^ (a >> 19);
    a = (a + 0x165667b1) + (a << 5);
    a = (a + 0xd3a2646c) ^ (a << 9);
    a = (a + 0xfd7046c5) + (a << 3);
    a = (a ^ 0xb55a4f09) ^ (a >> 16);
    return a;
}

__host__ __device__ thrust::default_random_engine random_engine(
        int iter, int index = 0, int depth = 0) {
    return thrust::default_random_engine(utilhash((index + 1) * iter) ^ utilhash(depth));
}

kdtree *initTree(kdtree *root){
	//postorder method to first get the left and right child on GPU Memory, then replace it with the memory on CPU, then copy the whole point to GPU
	if(root==nullptr) return nullptr;
	kdtree *dev_lc=initTree(root->lc);
	kdtree *dev_rc=initTree(root->rc);
	kdtree *tmp=new kdtree(root);
	tmp->lc=dev_lc;
	tmp->rc=dev_rc;
	kdtree *dev_root;
	hipMalloc(&dev_root,sizeof(kdtree));
	hipMemcpy(dev_root,tmp,sizeof(kdtree),hipMemcpyHostToDevice);
	return dev_root;
}

__global__ void updateStateKernel(GPUParticle *particle,glm::vec3 acc,float dt,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		particle[index].acc=acc;
		particle[index].vel+=dt*particle[index].acc;
		particle[index].newPos=particle[index].pos+dt*particle[index].vel;
		particle[index].index=0;
	}
}

__global__ void updateAccKernel(GPUParticle *particle,glm::vec3 acc,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		particle[index].acc=acc;
	}
}

__global__ void updateVelKernel(GPUParticle *particle,float dt,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		particle[index].vel+=dt*particle[index].acc;
	}
}

__global__ void updatePosKernel(GPUParticle *particle,float dt,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		particle[index].newPos=particle[index].pos+dt*particle[index].vel;
	}
}

__global__ void clearCellParticleKernel(GPUContainer *container,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		container->cell[index].index=0;
	}
}

__global__ void clearParticleNeighborKernel(GPUParticle *particle,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		particle[index].index=0;
	}
}

__global__ void putParticleInCellKernel(GPUParticle *particle,GPUContainer *container,int cell_particle_limit,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		glm::vec3 pos=particle[index].newPos-container->minPos;
		glm::vec3 cell_dim=container->cell_dim;
		pos.x=(int)(pos.x/container->cell_size);
		pos.y=(int)(pos.y/container->cell_size);
		pos.z=(int)(pos.z/container->cell_size);
		if(pos.x<0||pos.y<0||pos.z<0||pos.x>=cell_dim.x||pos.y>=cell_dim.y||pos.z>=cell_dim.z) return;
		int tmp=pos.x*cell_dim.y*cell_dim.z+pos.y*cell_dim.z+pos.z;
		//int value=container->cell[tmp].index;
		/*while(value!=atomicMax(&(container->cell[tmp].index),value)){
			value=container->cell[tmp].index;
		}*/
		/*
		this change is crucial
		*/
		int value=atomicAdd(&(container->cell[tmp].index),1);
		/*
		this change is crucial
		*/
		if(value>=cell_particle_limit){
			container->cell[tmp].index=cell_particle_limit;
			//return;
		}
		else{
			container->cell[tmp].particles[value]=index;
			particle[index].cellId=pos;
		}
	}
}

__global__ void findNeighborKernel(GPUParticle *particle,GPUContainer *container,int particle_neighbor_limit,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		glm::vec3 cell_dim=container->cell_dim;
		int x=particle[index].cellId.x;
		int y=particle[index].cellId.y;
		int z=particle[index].cellId.z;
		for(int i=x-1;i<=x+1;++i){
			for(int j=y-1;j<=y+1;++j){
				for(int k=z-1;k<=z+1;++k){
					if(i<0||j<0||k<0||i>=cell_dim.x||j>=cell_dim.y||k>=cell_dim.z) continue;
					int tmp=cell_dim.y*cell_dim.z*i+cell_dim.z*j+k;
					for(int t=0;t<container->cell[tmp].index;++t){
						//if(container->cell[tmp].particles[t]!=index)
							particle[index].neighbor[particle[index].index++]=container->cell[tmp].particles[t];
						if(particle[index].index>=particle_neighbor_limit) return;
					}//t
				}//k
			}//j
		}//i
	}
}

__device__ glm::vec3 spikyKernelOnGPU(float h,float PI,glm::vec3 r){
	float R=glm::length(r);
	float Epsilon=1e-4;
	if(R>h) return glm::vec3(0,0,0);
	glm::vec3 result=(float)(15.0/(PI*pow(h,3))*pow(h-R,2))*r/(R+Epsilon);
	return (float)(15.0/(PI*pow(h,6))*pow(h-R,2))*r/(R+Epsilon);
}

__device__ float poly6KernelOnGPU(float h,float PI,glm::vec3 r){
	float R=glm::length(r);
	if(R>h) return 0;
	return 315.0/(64.0*PI*pow(h,9))*pow((h*h-glm::length(r)*glm::length(r)),2);
}

__device__ void cubeContainerIntersectionOnGPU(GPUParticle &p,float radius,glm::vec3 dimension,float dt,float rebound_rest,int index){
	glm::vec3 pos=p.newPos;
	glm::vec3 lower_bound=-dimension/2.0f,upper_bound=-lower_bound;
	for(int i=0;i<3;++i){
		//thrust::default_random_engine rng = random_engine(index, i, index+i);
        //thrust::uniform_real_distribution<float> u01(0, 1);
		if(pos[i]-radius-lower_bound[i]<=0){
			p.newPos[i]+=(radius+lower_bound[i]-pos[i]);
			//p.newPos[i]+=(0.04*1.0*u01(rng)-0.02);
			p.newPos[i]+=(0.04*1.0);
			p.vel[i]*=-rebound_rest;
			p.newPos[i]+=p.vel[i]*dt/2;
		}
		if(pos[i]+radius-upper_bound[i]>=0){
			p.newPos[i]-=(pos[i]+radius-upper_bound[i]);
			//p.newPos[i]+=(0.04*1.0*u01(rng)-0.02);
			p.newPos[i]-=(0.04*1.0);
			p.vel[i]*=-rebound_rest;
			p.newPos[i]+=p.vel[i]*dt/2;
		}
	}
}

__device__ void cubePrimitiveIntersectionOnGPU(GPUParticle &p,GPUPrimitive &primitive,float radius){
	glm::vec3 center = primitive.pos;
	glm::vec3 diff = p.newPos - center;
	glm::vec3 normal;
	float colision_epsilon=1e-3;
	float xcollide,ycollide,zcollide;
	glm::vec3 m_hf_dims=primitive.cubeDim/2.0f;
	xcollide=(fabs(diff.x)-m_hf_dims[0]-radius-colision_epsilon);
	ycollide=(fabs(diff.y)-m_hf_dims[1]-radius-colision_epsilon);
	zcollide=(fabs(diff.z)-m_hf_dims[2]-radius-colision_epsilon);
	if(xcollide<0&&ycollide<0&&zcollide<0){
		if(xcollide>=ycollide&&xcollide>=zcollide){
			if(diff.x>0){
				normal=glm::vec3(1,0,0);
			}
			else{
				normal=glm::vec3(-1,0,0);
			}
			p.newPos-=normal*xcollide;
		}
		else if(ycollide>=xcollide&&ycollide>=zcollide){
			if(diff.y>0){
				normal=glm::vec3(0,1,0);
			}
			else{
				normal=glm::vec3(0,-1,0);
			}
			p.newPos-=normal*ycollide;
		}
		else if(zcollide>=ycollide&&zcollide>=xcollide){
			if(diff.z>0){
				normal=glm::vec3(0,0,1);
			}
			else{
				normal=glm::vec3(0,0,-1);
			}
			p.newPos-=normal*zcollide;
		}
	}
}

__device__ void spherePrimitiveIntersectionOnGPU(GPUParticle &p,GPUPrimitive &primitive,float radius){
	glm::vec3 pos=p.newPos;
	float dis=glm::length(pos-primitive.pos);
	if(primitive.sphereRadius+radius>dis){
		dis=primitive.sphereRadius+radius-dis;
		glm::vec3 nor=glm::normalize(pos-primitive.pos);
		p.newPos+=nor*dis;

		//p.newPos+=(float)(0.04*1.0*rand()/(RAND_MAX+1)-0.02)*nor;
		//p.newPos-=glm::dot(p.vel,nor)*nor*dt/2.0f;

		glm::vec3 tmp=p.vel-glm::dot(p.vel,nor)*nor;
		p.vel=-glm::dot(p.vel,nor)*nor+tmp;
	}
}

__device__ bool insideBoxOnGPU(glm::vec3 pos,kdtree *tree){
	if(pos.x<=tree->xMax&&pos.x>=tree->xMin&&pos.y<=tree->yMax&&pos.y>=tree->yMin&&
		pos.z<=tree->zMax&&pos.z>=tree->zMin){
		return true;
	}
	else return false;
}

__device__ void getNearbyTrianglesOnGPU(glm::vec3 pos,kdtree *tree, int *list){
	int count=0,num=0,n=0;
	kdtree *kd[1000];
	kd[count++]=tree;
	while(count<1000&&n!=count&&num<180){
		kdtree *current=kd[n];
		if(insideBoxOnGPU(pos,current)){
			if(current->lc==nullptr&&current->rc==nullptr) list[num++]=current->index;
			else{
				kd[count++]=current->lc;
				if(count>=1000) break;
				kd[count++]=current->rc;
			}
		}
		n++;
	}
}

__device__ glm::vec3 getNormalOnGPU(glm::vec3 *m_positions,glm::vec3 *m_normals,int *m_indices, unsigned short TriangleIndex){
	glm::vec3 n1,n2,n3,v1,v2,v3,n,crossN,v12,v13;
	unsigned int index1,index2,index3;
	index1=m_indices[3*TriangleIndex];
	index2=m_indices[3*TriangleIndex+1];
	index3=m_indices[3*TriangleIndex+2];
	v1=m_positions[index1];v2=m_positions[index2];v3=m_positions[index3];
	n1=m_normals[index1];n2=m_normals[index2];n3=m_normals[index3];
	
	v12=v1-v2;v13=v1-v3;
	v12=glm::normalize(v12);v13=glm::normalize(v13);
	crossN=glm::cross(v12,v13);
	crossN=glm::normalize(crossN);
	
	n=(n1+n2+n3);
	n=glm::normalize(n);

	if(glm::dot(n,crossN)<0) return -crossN;
	else return crossN;
}

__device__ float getDistanceOnGPU(glm::vec3 *m_positions,glm::vec3 *m_normals,int *m_indices,glm::vec3 p,unsigned short TriangleIndex){
	float dis,k,x;
	unsigned short index;
	index=m_indices[3*TriangleIndex];
	
	glm::vec3 normal=getNormalOnGPU(m_positions,m_normals,m_indices,TriangleIndex);
	
	glm::vec3 d=p-m_positions[index];
	x=(normal.x*d.x+normal.y*d.y+normal.z*d.z);
	return x;
}

__device__ void objmeshPrimitiveIntersectionOnGPU(GPUParticle &p,GPUPrimitive &primitive,float radius){
	float minDis=-1e7;
	float COLLISION_EPSILON=1e-2+radius;
	bool inCollision=false;
	glm::vec3 pos=p.newPos;
	glm::vec3 normal(0);
	int list[180];
	for(int i=0;i<180;++i) list[i]=-1;
	getNearbyTrianglesOnGPU(pos,primitive.mesh,list);
	pos-=primitive.pos;

	for(int i=0;i<180;i++){
		if(list[i]==-1) break;
		float tmp=getDistanceOnGPU(primitive.objVertex,primitive.objNormal,primitive.objIndices,pos,list[i])-COLLISION_EPSILON;
		if(tmp<0&&tmp>minDis&&tmp>-0.5){
			glm::vec3 n=getNormalOnGPU(primitive.objVertex,primitive.objNormal,primitive.objIndices,list[i]);
			normal=n;
			minDis=tmp;
			inCollision=true;
		}
	}
	if(inCollision){
		p.newPos-=normal*minDis;
		p.pos=p.newPos;
		glm::vec3 tmp=p.vel-glm::dot(p.vel,normal)*normal;
		p.vel=(tmp-glm::dot(p.vel,normal)*normal)*0.6f;
		p.newPos+=0.01f*p.vel/4.0f;
	}
	
}

__global__ void primitiveCollisionKernel(GPUParticle *particle,GPUPrimitive *primitive,int primitiveNum,float radius,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		for(int i=0;i<primitiveNum;++i){
			if(primitive[i].primitive_type==0){//cube
				cubePrimitiveIntersectionOnGPU(particle[index],primitive[i],radius);
			}
			else if(primitive[i].primitive_type==1){//sphere
				spherePrimitiveIntersectionOnGPU(particle[index],primitive[i],radius);
			}
			else if(primitive[i].primitive_type==2){//objmesh
				objmeshPrimitiveIntersectionOnGPU(particle[index],primitive[i],radius);
			}
		}
	}
}

__global__ void calculateLamdaKernel(GPUParticle *particle,float rest_density,float smooth_radius,float relaxation,float PI,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		float SumGrediant=0.0;
		float ParticleGrediant=0.0;
		float density=0.0;
		for(int i=0;i<particle[index].index;++i){
			int index1=particle[index].neighbor[i];
			float tmp=-glm::length(spikyKernelOnGPU(smooth_radius,PI,particle[index].newPos-particle[index1].newPos)/rest_density);
			ParticleGrediant+=tmp;
			SumGrediant+=pow(tmp,2);
			density+=poly6KernelOnGPU(smooth_radius,PI,particle[index].newPos-particle[index1].newPos);
		}
		SumGrediant += pow(ParticleGrediant,2);
	
		float densityConstriant=density/rest_density-1.0f;
		particle[index].lamda=-1*densityConstriant/(SumGrediant+relaxation);
	}
}

__global__ void PBDProjectionKernel(GPUParticle *particle,float rest_density,float smooth_radius,float PI,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		glm::vec3 dp=glm::vec3(0,0,0);
		for(int j=0;j<particle[index].index;++j){
			int index1=particle[index].neighbor[j];
			glm::vec3 tmp=particle[index].newPos-particle[index1].newPos;
			dp+=(particle[index].lamda+particle[index1].lamda)*spikyKernelOnGPU(smooth_radius,PI,particle[index].newPos-particle[index1].newPos)/rest_density;
		}
		particle[index].newPos=particle[index].newPos+dp;
			
		/*for(int j=0;j<s->pri.size();++j){
			s->pri[j]->staticIntersectionTest(s->particles[i]);
		}
		s->cont->staticIntersection(s->particles[i]);*/
	}
}

__global__ void containerCollisionKernel(GPUParticle *particle,GPUContainer *container,float particle_radius,float dt,float rebound_rest,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		if(container->container_type==0){//CUBE
			cubeContainerIntersectionOnGPU(particle[index],particle_radius,container->cubeDim,dt,rebound_rest,index);
		}
	}
}

__global__ void calculateNewVelocityKernel(GPUParticle *particle,float dt,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		particle[index].vel=(particle[index].newPos-particle[index].pos)/dt;
		particle[index].pos=particle[index].newPos;
	}
}

void initParticleOnGPU(glm::vec3 particleDim,glm::vec3 particleMinPos,float particleDis){
	GparticleDim=particleDim;
	GparticleNum=GparticleDim[0]*GparticleDim[1]*GparticleDim[2];
	Gparticles=new GPUParticle[GparticleNum];
	for(int i=0;i<GparticleDim.x;++i){
		for(int j=0;j<GparticleDim.y;++j){
			for(int k=0;k<GparticleDim.z;++k){
				int index=i*GparticleDim.y*GparticleDim.z+j*GparticleDim.z+k;
				Gparticles[index].acc=glm::vec3(0);
				Gparticles[index].vel=glm::vec3(0);
				Gparticles[index].pos=particleMinPos+particleDis*glm::vec3(i,j,k);
				Gparticles[index].newPos=glm::vec3(0);
				Gparticles[index].cellId=glm::vec3(0);
				Gparticles[index].lamda=0;
				Gparticles[index].mass=0;
				Gparticles[index].index=0;
			}//k
		}//j
	}
	hipMalloc(&dev_particles,GparticleNum*sizeof(GPUParticle));
	hipMemcpy(dev_particles,Gparticles,GparticleNum*sizeof(GPUParticle),hipMemcpyHostToDevice);
}

void initCubeContainerOnGPU(glm::vec3 dimension,float cell_size){
	Gcontainer=new GPUContainer();
	Gcontainer->container_type=CONTAINER::CONTAINER_CUBE;
	Gcontainer->cell_size=cell_size;
	Gcontainer->cubeDim=dimension;
	Gcontainer->boundingBox=dimension;
	Gcontainer->minPos=-dimension/2.0f;
	int x=(int)ceil(Gcontainer->boundingBox.x/cell_size);
	int y=(int)ceil(Gcontainer->boundingBox.y/cell_size);
	int z=(int)ceil(Gcontainer->boundingBox.z/cell_size);
	Gcontainer->cell_dim=glm::vec3(x,y,z);

	initCellOnGPU(Gcontainer->boundingBox,Gcontainer->cell_dim,cell_size);
	Gcontainer->cell=dev_cell;

	hipMalloc(&dev_container,sizeof(GPUContainer));
	hipMemcpy(dev_container,Gcontainer,sizeof(GPUContainer),hipMemcpyHostToDevice);
}

void initSphereContainerOnGPU(float radius,float cell_size){
	Gcontainer=new GPUContainer();
	Gcontainer->container_type=CONTAINER::CONTAINER_SPHERE;
	Gcontainer->cell_size=cell_size;
	Gcontainer->sphereRadius=radius;
	Gcontainer->boundingBox=2.0f*glm::vec3(radius);
	Gcontainer->minPos=glm::vec3(-radius);
	int x=(int)ceil(Gcontainer->boundingBox.x/cell_size);
	int y=(int)ceil(Gcontainer->boundingBox.y/cell_size);
	int z=(int)ceil(Gcontainer->boundingBox.z/cell_size);
	Gcontainer->cell_dim=glm::vec3(x,y,z);

	initCellOnGPU(Gcontainer->boundingBox,Gcontainer->cell_dim,cell_size);
	Gcontainer->cell=dev_cell;

	hipMalloc(&dev_container,sizeof(GPUContainer));
	hipMemcpy(dev_container,Gcontainer,sizeof(GPUContainer),hipMemcpyHostToDevice);
}

void initCellOnGPU(glm::vec3 boundingBox,glm::vec3 cell_dim,float cell_size){
	int x=cell_dim.x;
	int y=cell_dim.y;
	int z=cell_dim.z;
	Gcell=new GPUCell[x*y*z];
	for(int i=0;i<x;++i){
		for(int j=0;j<y;++j){
			for(int k=0;k<z;++k){
				int index=i*y*z+j*z+k;
				Gcell[index].pos=-boundingBox/2.0f+glm::vec3(i,j,k)*cell_size;
				Gcell[index].index=0;
			}//k
		}//j
	}//i
	hipMalloc(&dev_cell,x*y*z*sizeof(GPUCell));
	hipMemcpy(dev_cell,Gcell,x*y*z*sizeof(GPUCell),hipMemcpyHostToDevice);
}

void initPrimitiveOnGPU(GPUPrimitive *primitive,int num){
	Gprimitive=primitive;
	GprimitiveNum=num;
	hipMalloc(&dev_primitive,num*sizeof(GPUPrimitive));
	hipMemcpy(dev_primitive,Gprimitive,num*sizeof(GPUPrimitive),hipMemcpyHostToDevice);
}

void updateOnGPU(){
	//updateAccOnGPU();
	//updateVelOnGPU();
	//updatePosOnGPU();
	//clearParticleNeighborOnGPU();
	updateStateOnGPU();

	clearCellParticleOnGPU();
	putParticleInCellOnGPU();
	findNeighborOnGPU();
	for(int i=0;i<4;++i){
		calculateLamdaOnGPU();
		PBDProjectionOnGPU();
		primitiveCollisionOnGPU();
		containerCollisionOnGPU();
		//hipMemcpy(Gparticles,dev_particles,GparticleNum*sizeof(GPUParticle),hipMemcpyDeviceToHost);
		//cout<<"newPos "<<Gparticles[0].newPos.x<<","<<Gparticles[0].newPos.y<<","<<Gparticles[0].newPos.z<<endl;
		//cout<<"vel "<<Gparticles[0].vel.x<<","<<Gparticles[0].vel.y<<","<<Gparticles[0].vel.z<<endl;
	}
	calculateNewVelOnGPU();
	hipMemcpy(Gparticles,dev_particles,GparticleNum*sizeof(GPUParticle),hipMemcpyDeviceToHost);
}

void updateAccOnGPU(){
	updateAccKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,gravityVector,GparticleNum);
}

void updateVelOnGPU(){
	updateVelKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dt,GparticleNum);
}

void updatePosOnGPU(){
	updatePosKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dt,GparticleNum);
}

void putParticleInCellOnGPU(){
	putParticleInCellKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dev_container,GPU_cell_particle_limit,GparticleNum);
	//hipMemcpy(Gcontainer,dev_container,sizeof(GPUContainer),hipMemcpyDeviceToHost);
	//hipMemcpy(Gcell,dev_cell,20*20*20*sizeof(GPUCell),hipMemcpyDeviceToHost);
	/*for(int i=0;i<20*20*20;++i){
		if(Gcell[i].index!=0){
			//for(int j=0;j<Gcell[i].index;++j)
				//cout<<Gcell[i].particles[j]<<","<<i<<endl;
			cout<<Gcell[i].index<<endl;
		}
	}*/
}

void clearCellParticleOnGPU(){
	int num=Gcontainer->cell_dim.x*Gcontainer->cell_dim.y*Gcontainer->cell_dim.z;
	clearCellParticleKernel<<<(num+255)/256,256>>>(dev_container,num);
}

void clearParticleNeighborOnGPU(){
	clearParticleNeighborKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,GparticleNum);
}

void findNeighborOnGPU(){
	findNeighborKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dev_container,GPU_particle_neighbor_limit,GparticleNum);
	//hipMemcpy(Gparticles,dev_particles,GparticleNum*sizeof(GPUParticle),hipMemcpyDeviceToHost);
	/*for(int i=0;i<GparticleNum;++i){
		cout<<Gparticles[i].index<<","<<i<<endl;
	}
	cout<<"@"<<endl;*/
}

void calculateLamdaOnGPU(){
	calculateLamdaKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,rest_density,smooth_radius,relaxation,PI,GparticleNum);
}

void PBDProjectionOnGPU(){
	PBDProjectionKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,rest_density,smooth_radius,PI,GparticleNum);
}

void calculateNewVelOnGPU(){
	calculateNewVelocityKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dt,GparticleNum);
}

void containerCollisionOnGPU(){
	containerCollisionKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dev_container,particle_radius,dt,rebound_rest,GparticleNum);
}

void primitiveCollisionOnGPU(){
	/*glm::vec3 *tmp,*dev_tmp;
	float *tmp1,*dev_tmp1;
	tmp=new glm::vec3[1];
	tmp1=new float[1];
	hipMalloc(&dev_tmp,sizeof(glm::vec3));
	hipMalloc(&dev_tmp1,sizeof(float));*/
	primitiveCollisionKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,dev_primitive,GprimitiveNum,particle_radius,GparticleNum);
	/*hipMemcpy(tmp,dev_tmp,sizeof(glm::vec3),hipMemcpyDeviceToHost);
	hipMemcpy(tmp1,dev_tmp1,sizeof(float),hipMemcpyDeviceToHost);
	cout<<tmp[0].x<<","<<tmp[0].y<<","<<tmp[0].z<<endl;
	cout<<tmp1[0]<<endl;
	delete(tmp);
	delete(tmp1);*/
}

glm::vec3 getPos(int n){
	return Gparticles[n].pos;
}

void updateStateOnGPU(){
	updateStateKernel<<<(GparticleNum+255)/256,256>>>(dev_particles,gravityVector,dt,GparticleNum);
}